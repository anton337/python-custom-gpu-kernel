
#include <hip/hip_runtime.h>
extern "C" __global__ void sum ( const float *A
                               , const float *B
                               ,       float *C
                               ,         int  N
    )
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<N) {
    C[i] = A[i] + B[i];
  }
}
